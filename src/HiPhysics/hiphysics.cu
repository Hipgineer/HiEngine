#include "hiphysics.h"
#include "hiphysicsPBD.h"

HiPhysicsUPtr HiPhysics::Create() {
    auto solver = HiPhysicsUPtr(new HiPhysics());
    if(!solver->Init())
        return nullptr;
    return std::move(solver);
}

bool HiPhysics::ClearMemory() {

    hipError_t hipError_t;

    hipFree(dm_DataFluid.colorValues);
    hipFree(dm_DataFluid.positions);
    hipFree(dm_DataFluid.velocities);
    hipFree(dm_DataFluid.phases);
    hipFree(dm_DataFluid.constraints);
    hipFree(dm_DataFluid.lambdas);
    hipFree(dm_DataFluid.correctedPos);
    hipFree(dm_DataFluid.deltaPos);
    hipFree(dm_DataFluid.gridIndices);
    hipFree(dm_DataFluid.numPartInGrids);
    hipFree(dm_DataFluid.nearGridID);
    hipFree(dm_DataFluid.commonParam);
    hipFree(dm_DataFluid.phaseParam);

    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Fail to HiPhysics::ClearMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
    return true;

}

bool HiPhysics::SetMemory(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();

    //TODO : Dynamic allocation!
    // the number of particles is varying during the simulations!!
    hipMalloc(&(dm_DataFluid.colorValues), count*sizeof(float));
    hipMemset(dm_DataFluid.colorValues, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.colorValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.positions, count*sizeof(glm::vec3));
	hipMemcpy(dm_DataFluid.positions, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.positions %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.velocities, count*sizeof(glm::vec3));
	hipMemcpy(dm_DataFluid.velocities, simBuffer->m_velocities.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.velocities %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.phases, count*sizeof(int32_t));
	hipMemcpy(dm_DataFluid.phases, simBuffer->m_phases.data(), count*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.phases %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.constraints, count*sizeof(float));
    hipMemset(dm_DataFluid.constraints, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.constraints %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.lambdas, count*sizeof(float));
    hipMemset(dm_DataFluid.lambdas, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.lambdas %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.correctedPos, count*sizeof(glm::vec3));
    hipMemset(dm_DataFluid.correctedPos, 0, count*sizeof(glm::vec3));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.correctedPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.deltaPos, count*sizeof(glm::vec3));
    hipMemset(dm_DataFluid.deltaPos, 0, count*sizeof(glm::vec3));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.deltaPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.gridIndices, count*sizeof(int32_t));
    hipMemset(dm_DataFluid.gridIndices, 0, count*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.gridIndices %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    // TODO
    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);
    float H = simBuffer->m_commonParam.radius * 1.2f * 2.0f * 2.0f;
    int32_t ix = static_cast<int32_t>((maxPosition.x - (simBuffer->m_commonParam.radius) - minPosition.x)/H)+1;
    int32_t iy = static_cast<int32_t>((maxPosition.y - (simBuffer->m_commonParam.radius) - minPosition.y)/H)+1;
    int32_t iz = static_cast<int32_t>((maxPosition.z - (simBuffer->m_commonParam.radius) - minPosition.z)/H)+1;
    hipMalloc(&dm_DataFluid.numPartInGrids, 1000*(ix)*(iy)*(iz)*sizeof(int32_t));
    hipMemset(dm_DataFluid.numPartInGrids, 0, 1000*(ix)*(iy)*(iz)*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.numPartInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
    hipMalloc(&dm_DataFluid.nearGridID, 3*(ix+1)*(iy+1)*(iz+1)*27*sizeof(int32_t));
    hipMemset(dm_DataFluid.nearGridID, 0, 3*(ix+1)*(iy+1)*(iz+1)*27*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.nearGridID %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.commonParam, sizeof(CommonParameters));
	hipMemcpy(dm_DataFluid.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataFluid.phaseParam, simBuffer->m_phaseParam.size()*sizeof(PhaseParameters));
	hipMemcpy(dm_DataFluid.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataFluid.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}


    hipMalloc(&dm_SimParameters.commonParam, sizeof(CommonParameters));
	hipMemcpy(dm_SimParameters.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_SimParameters.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_SimParameters.phaseParam, simBuffer->m_phaseParam.size()*sizeof(PhaseParameters));
	hipMemcpy(dm_SimParameters.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_SimParameters.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::SetMemoryCloth(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();
    uint64_t nStretchLines = simBuffer->GetNumStretchLines();
    uint64_t nBendLines = simBuffer->GetNumBendLines();
    uint64_t nShearLines = simBuffer->GetNumShearLines();
    uint64_t nTriangles = simBuffer->GetNumTriangles();

    //TODO : Dynamic allocation!
    // the number of particles is varying during the simulations!!
    hipMalloc(&(dm_DataCloth.colorValues), count*sizeof(float));
    hipMemset(dm_DataCloth.colorValues, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.colorValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.positions, count*sizeof(glm::vec3));
	hipMemcpy(dm_DataCloth.positions, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.positions %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.velocities, count*sizeof(glm::vec3));
	hipMemcpy(dm_DataCloth.velocities, simBuffer->m_velocities.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.velocities %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.phases, count*sizeof(int32_t));
	hipMemcpy(dm_DataCloth.phases, simBuffer->m_phases.data(), count*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.phases %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.stretchID, 2*nStretchLines*sizeof(int32_t));
	hipMemcpy(dm_DataCloth.stretchID, simBuffer->m_stretchID.data(), 2*nStretchLines*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.stretchID %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.bendID, 2*nBendLines*sizeof(int32_t));
    hipMemcpy(dm_DataCloth.bendID, simBuffer->m_bendID.data(), 2*nBendLines*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.bendID %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.shearID, nShearLines*sizeof(int32_t));
    hipMemcpy(dm_DataCloth.shearID, simBuffer->m_shearID.data(), nShearLines*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.shearID %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.correctedPos, count*sizeof(glm::vec3));
	hipMemcpy(dm_DataCloth.correctedPos, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.correctedPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_DataCloth.deltaPos, count*sizeof(glm::vec3));
    hipMemset(dm_DataCloth.deltaPos, 0, count*sizeof(glm::vec3));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_DataCloth.deltaPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_SimParameters.commonParam, sizeof(CommonParameters));
	hipMemcpy(dm_SimParameters.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_SimParameters.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_SimParameters.phaseParam, simBuffer->m_phaseParam.size()*sizeof(PhaseParameters));
	hipMemcpy(dm_SimParameters.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_SimParameters.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::GetMemory(SimBufferPtr simBuffer) {
    hipError_t hipError_t;

    uint64_t count = simBuffer->GetNumParticles();

	hipMemcpy(simBuffer->m_colorValues.data(), dm_DataFluid.colorValues, count*sizeof(float),    hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_positions.data(),   dm_DataFluid.positions,   count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_velocities.data(),  dm_DataFluid.velocities,  count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_phases.data(),      dm_DataFluid.phases,      count*sizeof(int32_t),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::Init () {   
    // cudamemcopy

    return true;
}

void HiPhysics::UpdateSolver(SimBufferPtr simBuffer) {
    m_numParticles = simBuffer->GetNumParticles();
    
    if (m_numParticles > 0)
    {
        /// APPLY THE CHANGE BY USER INTERFACE
        MemsetFromHost(simBuffer);

        /// 
        PredictPosition(simBuffer);

        
        for (int32_t ii = 0; ii < simBuffer->m_commonParam.iterationNumber; ++ii)
        {
        /// COMPUTE GRID INDEX COUNT THE NUMBER OF PARTICLES IN THE GRID
            ComputeGridIndices(simBuffer);

        /// SORT BY GRID INDEX
            SortVariablesByIndices(simBuffer);

        /// COMPUTE CONSTRAINTS
            ComputeConstraint(simBuffer);
        }
            

        /// UPDATE PARTICLE POSITIONS
        UpdateVelPos(simBuffer);

        /// GET VALUES FOR RENDERING PARTICLE COLOR
        GetRenderingVariable(simBuffer);
    }
}

bool HiPhysics::MemsetFromHost(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();

	// hipMemcpy(dm_DataFluid.positions, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_DataFluid.positions %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }

	// hipMemcpy(dm_DataFluid.velocities, simBuffer->m_velocities.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_DataFluid.velocities %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }

	// hipMemcpy(dm_DataFluid.phases, simBuffer->m_phases.data(), count*sizeof(int32_t), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_DataFluid.phases %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }
    
	hipMemcpy(dm_DataFluid.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_DataFluid.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

	hipMemcpy(dm_DataFluid.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_DataFluid.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::PredictPosition(SimBufferPtr simBuffer) {

    hipError_t hipError_t; // TODO : make it as a member variable.

    // 0. Predict Position
    kePredictPosition<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::kePredictPosition :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    return true;
}

bool HiPhysics::ComputeGridIndices(SimBufferPtr simBuffer){

    hipError_t hipError_t; // TODO : make it as a member variable.

    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);
    float H = simBuffer->m_commonParam.radius * 1.2f * 2.0f * 2.0f;
    int32_t ix = static_cast<int32_t>((maxPosition.x - (simBuffer->m_commonParam.radius) - minPosition.x)/H)+1;
    int32_t iy = static_cast<int32_t>((maxPosition.y - (simBuffer->m_commonParam.radius) - minPosition.y)/H)+1;
    int32_t iz = static_cast<int32_t>((maxPosition.z - (simBuffer->m_commonParam.radius) - minPosition.z)/H)+1;
    hipMemset(dm_DataFluid.numPartInGrids, 0, (ix)*(iy)*(iz)*sizeof(int32_t));
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("clear(Memset) dm_DataFluid.numPartInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // 1. assign Grid ID to Particles.
    keComputeGridID<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keComputeGridID %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();


    // 2. Count the number of Particles in each Grids.
    keCountParticlesInGrids<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keCountParticlesInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // 3. Inclusive scan the number of particels in each Grids.
    {
        thrust::device_vector<int32_t> temp(dm_DataFluid.numPartInGrids,dm_DataFluid.numPartInGrids + ix*iy*iz);
        thrust::device_ptr<int32_t> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.numPartInGrids);
        thrust::inclusive_scan(temp.begin(), temp.end(), dev_ptr);
    }

    return true;
}

bool HiPhysics::SortVariablesByIndices(SimBufferPtr simBuffer) {
    
    thrust::device_vector<int> indices(m_numParticles); 
    thrust::sequence(indices.begin(),indices.end());
    thrust::sort_by_key(dm_DataFluid.gridIndices,dm_DataFluid.gridIndices+m_numParticles,indices.begin());

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.colorValues);
        thrust::device_vector<float> temp(dm_DataFluid.colorValues,dm_DataFluid.colorValues+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.positions);
        thrust::device_vector<glm::vec3> temp(dm_DataFluid.positions,dm_DataFluid.positions+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }
    
    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.velocities);
        thrust::device_vector<glm::vec3> temp(dm_DataFluid.velocities,dm_DataFluid.velocities+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<int32_t> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.phases);
        thrust::device_vector<int32_t> temp(dm_DataFluid.phases,dm_DataFluid.phases+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.constraints);
        thrust::device_vector<float> temp(dm_DataFluid.constraints,dm_DataFluid.constraints+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.lambdas);
        thrust::device_vector<float> temp(dm_DataFluid.lambdas,dm_DataFluid.lambdas+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }
    
    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.deltaPos);
        thrust::device_vector<glm::vec3> temp(dm_DataFluid.deltaPos,dm_DataFluid.deltaPos+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_DataFluid.correctedPos);
        thrust::device_vector<glm::vec3> temp(dm_DataFluid.correctedPos,dm_DataFluid.correctedPos+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    return true;
}

bool HiPhysics::ComputeConstraint(SimBufferPtr simBuffer){

    hipError_t hipError_t;

    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);

    // Compute Constraints
    keComputeConstraint<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputeConstraint  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // Correct Positions
    keComputePositionCorrection<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputePositionCorrection  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // Update Corrected Positions
    keUpdateCorretedPosition<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keUpdatePosition :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

	hipMemcpy(simBuffer->m_positions.data(),      dm_DataFluid.correctedPos,      m_numParticles*sizeof(glm::vec3),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::ComputeConstraint-memcpyDelPos :%s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::UpdateVelPos(SimBufferPtr simBuffer){

    hipError_t hipError_t; // TODO : 맴버변수화 

    keUpdateVelPos<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::UpdateVelPos :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}

bool HiPhysics::GetRenderingVariable(SimBufferPtr simBuffer){

    hipError_t hipError_t;

    keGetRenderValues<<< 1 +  m_numParticles/256, 256>>>(dm_DataFluid, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keGetRenderValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}










void HiPhysics::UpdateSolverCloth(SimBufferPtr simBuffer) {
    m_numParticles = simBuffer->GetNumParticles();
    
    if (m_numParticles > 0)
    {
        /// APPLY THE CHANGE BY USER INTERFACE
        MemsetFromHostCloth(simBuffer);

        /// 
        PredictPositionCloth(simBuffer);

        ///
        
        for (int32_t ii = 0; ii < simBuffer->m_commonParam.iterationNumber; ++ii)
        {
            ComputeConstraintCloth(simBuffer);
        }
            
        /// UPDATE PARTICLE POSITIONS
        UpdateVelPosCloth(simBuffer);

        /// GET VALUES FOR RENDERING PARTICLE COLOR
        GetRenderingVariableCloth(simBuffer);
    }
}


bool HiPhysics::MemsetFromHostCloth(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();
    
	hipMemcpy(dm_SimParameters.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_SimParameters.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

	hipMemcpy(dm_SimParameters.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_SimParameters.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::PredictPositionCloth(SimBufferPtr simBuffer) {

    hipError_t hipError_t; // TODO : make it as a member variable.

    // 0. Predict Position
    kePredictPositionCloth<<< 1 +  m_numParticles/256, 256>>>(dm_DataCloth, dm_SimParameters, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::kePredictPosition :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    return true;
}


bool HiPhysics::ComputeConstraintCloth(SimBufferPtr simBuffer){

    hipError_t hipError_t;
    // Compute Constraints
    keComputeStretchCloth<<< 1 +  simBuffer->GetNumStretchLines()/256, 256>>>(dm_DataCloth, dm_SimParameters, simBuffer->GetNumStretchLines());
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputeConstraintCloth  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    
    keComputeBendCloth<<< 1 +  simBuffer->GetNumBendLines()/256, 256>>>(dm_DataCloth, dm_SimParameters, simBuffer->GetNumBendLines());
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputeConstraintCloth  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    
    // keComputeShearCloth<<< 1 +  simBuffer->GetNumShearLines()/256, 256>>>(dm_DataCloth, dm_SimParameters, simBuffer->GetNumShearLines());
    // hipError_t = hipGetLastError();
    // if (hipError_t != hipSuccess)
    // {
    //     printf("Error at HiPhysics::ComputeConstraint-keComputeConstraintCloth  %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    // }
    // hipDeviceSynchronize();

    // Update Corrected Positions
    keUpdateCorretedPositionCloth<<< 1 +  m_numParticles/256, 256>>>(dm_DataCloth, dm_SimParameters, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keUpdateCorretedPositionCloth :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

	// hipMemcpy(simBuffer->m_positions.data(),      dm_DataCloth.correctedPos,      m_numParticles*sizeof(glm::vec3),  hipMemcpyDeviceToHost);
	// hipDeviceSynchronize(); hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Error at HiPhysics::ComputeConstraint-memcpyDelPos :%s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }

    return true;
}

bool HiPhysics::UpdateVelPosCloth(SimBufferPtr simBuffer){

    hipError_t hipError_t; // TODO : 맴버변수화 

    keUpdateVelPosCloth<<< 1 +  m_numParticles/256, 256>>>(dm_DataCloth, dm_SimParameters, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::UpdateVelPos :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}

bool HiPhysics::GetRenderingVariableCloth(SimBufferPtr simBuffer){

    hipError_t hipError_t;

    keGetRenderValuesCloth<<< 1 +  m_numParticles/256, 256>>>(dm_DataCloth, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keGetRenderValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}



bool HiPhysics::GetMemoryCloth(SimBufferPtr simBuffer) {
    hipError_t hipError_t;

    uint64_t count = simBuffer->GetNumParticles();

	hipMemcpy(simBuffer->m_colorValues.data(), dm_DataCloth.colorValues, count*sizeof(float),    hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_positions.data(),   dm_DataCloth.positions,   count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_velocities.data(),  dm_DataCloth.velocities,  count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_phases.data(),      dm_DataCloth.phases,      count*sizeof(int32_t),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}
