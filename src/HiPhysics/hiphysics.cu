#include "hiphysics.h"
#include "hiphysicsPBD.h"

HiPhysicsUPtr HiPhysics::Create() {
    auto solver = HiPhysicsUPtr(new HiPhysics());
    if(!solver->Init())
        return nullptr;
    return std::move(solver);
}

bool HiPhysics::ClearMemory() {

    hipError_t hipError_t;

    hipFree(dm_Data.colorValues);
    hipFree(dm_Data.positions);
    hipFree(dm_Data.velocities);
    hipFree(dm_Data.phases);
    hipFree(dm_Data.constraints);
    hipFree(dm_Data.lambdas);
    hipFree(dm_Data.correctedPos);
    hipFree(dm_Data.deltaPos);
    hipFree(dm_Data.gridIndices);
    hipFree(dm_Data.numPartInGrids);
    hipFree(dm_Data.nearGridID);
    hipFree(dm_Data.commonParam);
    hipFree(dm_Data.phaseParam);

    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Fail to HiPhysics::ClearMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
    return true;

}

bool HiPhysics::SetMemory(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();

    //TODO : Dynamic allocation!
    // the number of particles is varying during the simulations!!
    hipMalloc(&(dm_Data.colorValues), count*sizeof(float));
    hipMemset(dm_Data.colorValues, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.colorValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.positions, count*sizeof(glm::vec3));
	hipMemcpy(dm_Data.positions, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.positions %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.velocities, count*sizeof(glm::vec3));
	hipMemcpy(dm_Data.velocities, simBuffer->m_velocities.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.velocities %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.phases, count*sizeof(int32_t));
	hipMemcpy(dm_Data.phases, simBuffer->m_phases.data(), count*sizeof(int32_t), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.phases %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.constraints, count*sizeof(float));
    hipMemset(dm_Data.constraints, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.constraints %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.lambdas, count*sizeof(float));
    hipMemset(dm_Data.lambdas, 0, count*sizeof(float));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.lambdas %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.correctedPos, count*sizeof(glm::vec3));
    hipMemset(dm_Data.correctedPos, 0, count*sizeof(glm::vec3));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.correctedPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.deltaPos, count*sizeof(glm::vec3));
    hipMemset(dm_Data.deltaPos, 0, count*sizeof(glm::vec3));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.deltaPos %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.gridIndices, count*sizeof(int32_t));
    hipMemset(dm_Data.gridIndices, 0, count*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.gridIndices %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    // TODO
    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);
    float H = simBuffer->m_commonParam.radius * 1.2f * 2.0f * 2.0f;
    int32_t ix = static_cast<int32_t>((maxPosition.x - (simBuffer->m_commonParam.radius) - minPosition.x)/H)+1;
    int32_t iy = static_cast<int32_t>((maxPosition.y - (simBuffer->m_commonParam.radius) - minPosition.y)/H)+1;
    int32_t iz = static_cast<int32_t>((maxPosition.z - (simBuffer->m_commonParam.radius) - minPosition.z)/H)+1;
    hipMalloc(&dm_Data.numPartInGrids, 1000*(ix)*(iy)*(iz)*sizeof(int32_t));
    hipMemset(dm_Data.numPartInGrids, 0, 1000*(ix)*(iy)*(iz)*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.numPartInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
    hipMalloc(&dm_Data.nearGridID, 3*(ix+1)*(iy+1)*(iz+1)*27*sizeof(int32_t));
    hipMemset(dm_Data.nearGridID, 0, 3*(ix+1)*(iy+1)*(iz+1)*27*sizeof(int32_t));
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.nearGridID %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.commonParam, sizeof(CommonParameters));
	hipMemcpy(dm_Data.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    hipMalloc(&dm_Data.phaseParam, simBuffer->m_phaseParam.size()*sizeof(PhaseParameters));
	hipMemcpy(dm_Data.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("MallocMemcpy dm_Data.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::GetMemory(SimBufferPtr simBuffer) {
    hipError_t hipError_t;

    uint64_t count = simBuffer->GetNumParticles();

	hipMemcpy(simBuffer->m_colorValues.data(), dm_Data.colorValues, count*sizeof(float),    hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_positions.data(),   dm_Data.positions,   count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_velocities.data(),  dm_Data.velocities,  count*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}
	hipMemcpy(simBuffer->m_phases.data(),      dm_Data.phases,      count*sizeof(int32_t),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::GetMemory %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::Init () {   
    // cudamemcopy

    return true;
}

void HiPhysics::UpdateSolver(SimBufferPtr simBuffer) {
    m_numParticles = simBuffer->GetNumParticles();
    
    if (m_numParticles > 0)
    {
        /// APPLY THE CHANGE BY USER INTERFACE
        MemsetFromHost(simBuffer);

        /// 
        PredictPosition(simBuffer);

        
        for (int32_t ii = 0; ii < simBuffer->m_commonParam.iterationNumber; ++ii)
        {
        /// COMPUTE GRID INDEX COUNT THE NUMBER OF PARTICLES IN THE GRID
            ComputeGridIndices(simBuffer);

        /// SORT BY GRID INDEX
            SortVariablesByIndices(simBuffer);

        /// COMPUTE CONSTRAINTS
            ComputeConstraint(simBuffer);
        }
            

        /// UPDATE PARTICLE POSITIONS
        UpdateVelPos(simBuffer);

        /// GET VALUES FOR RENDERING PARTICLE COLOR
        GetRenderingVariable(simBuffer);
    }
}

bool HiPhysics::MemsetFromHost(SimBufferPtr simBuffer) {
    hipError_t hipError_t;
    uint64_t count = simBuffer->GetNumParticles();

	// hipMemcpy(dm_Data.positions, simBuffer->m_positions.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_Data.positions %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }

	// hipMemcpy(dm_Data.velocities, simBuffer->m_velocities.data(), count*sizeof(glm::vec3), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_Data.velocities %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }

	// hipMemcpy(dm_Data.phases, simBuffer->m_phases.data(), count*sizeof(int32_t), hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
    // hipError_t = hipGetLastError();
	// if (hipError_t != hipSuccess)
  	// {
    //     printf("Memcpy dm_Data.phases %s\n",hipGetErrorString(hipError_t));
    //     exit(1);
    //     return false;
  	// }
    
	hipMemcpy(dm_Data.commonParam, &simBuffer->m_commonParam, sizeof(CommonParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_Data.commonParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

	hipMemcpy(dm_Data.phaseParam, simBuffer->m_phaseParam.data(), simBuffer->m_phaseParam.size()*sizeof(PhaseParameters), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
    hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Memcpy dm_Data.phaseParam %s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::PredictPosition(SimBufferPtr simBuffer) {

    hipError_t hipError_t; // TODO : make it as a member variable.

    // 0. Predict Position
    kePredictPosition<<< 1 +  m_numParticles/256, 256>>>(dm_Data, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::kePredictPosition :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    return true;
}

bool HiPhysics::ComputeGridIndices(SimBufferPtr simBuffer){

    hipError_t hipError_t; // TODO : make it as a member variable.

    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);
    float H = simBuffer->m_commonParam.radius * 1.2f * 2.0f * 2.0f;
    int32_t ix = static_cast<int32_t>((maxPosition.x - (simBuffer->m_commonParam.radius) - minPosition.x)/H)+1;
    int32_t iy = static_cast<int32_t>((maxPosition.y - (simBuffer->m_commonParam.radius) - minPosition.y)/H)+1;
    int32_t iz = static_cast<int32_t>((maxPosition.z - (simBuffer->m_commonParam.radius) - minPosition.z)/H)+1;
    hipMemset(dm_Data.numPartInGrids, 0, (ix)*(iy)*(iz)*sizeof(int32_t));
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("clear(Memset) dm_Data.numPartInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // 1. assign Grid ID to Particles.
    keComputeGridID<<< 1 +  m_numParticles/256, 256>>>(dm_Data, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keComputeGridID %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();


    // 2. Count the number of Particles in each Grids.
    keCountParticlesInGrids<<< 1 +  m_numParticles/256, 256>>>(dm_Data, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keCountParticlesInGrids %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // 3. Inclusive scan the number of particels in each Grids.
    {
        thrust::device_vector<int32_t> temp(dm_Data.numPartInGrids,dm_Data.numPartInGrids + ix*iy*iz);
        thrust::device_ptr<int32_t> dev_ptr = thrust::device_pointer_cast(dm_Data.numPartInGrids);
        thrust::inclusive_scan(temp.begin(), temp.end(), dev_ptr);
    }

    return true;
}

bool HiPhysics::SortVariablesByIndices(SimBufferPtr simBuffer) {
    
    thrust::device_vector<int> indices(m_numParticles); 
    thrust::sequence(indices.begin(),indices.end());
    thrust::sort_by_key(dm_Data.gridIndices,dm_Data.gridIndices+m_numParticles,indices.begin());

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_Data.colorValues);
        thrust::device_vector<float> temp(dm_Data.colorValues,dm_Data.colorValues+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_Data.positions);
        thrust::device_vector<glm::vec3> temp(dm_Data.positions,dm_Data.positions+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }
    
    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_Data.velocities);
        thrust::device_vector<glm::vec3> temp(dm_Data.velocities,dm_Data.velocities+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<int32_t> dev_ptr = thrust::device_pointer_cast(dm_Data.phases);
        thrust::device_vector<int32_t> temp(dm_Data.phases,dm_Data.phases+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_Data.constraints);
        thrust::device_vector<float> temp(dm_Data.constraints,dm_Data.constraints+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dm_Data.lambdas);
        thrust::device_vector<float> temp(dm_Data.lambdas,dm_Data.lambdas+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }
    
    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_Data.deltaPos);
        thrust::device_vector<glm::vec3> temp(dm_Data.deltaPos,dm_Data.deltaPos+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    {
        thrust::device_ptr<glm::vec3> dev_ptr = thrust::device_pointer_cast(dm_Data.correctedPos);
        thrust::device_vector<glm::vec3> temp(dm_Data.correctedPos,dm_Data.correctedPos+m_numParticles);
        thrust::gather(indices.begin(),indices.end(), temp.data(), dev_ptr);
    }

    return true;
}

bool HiPhysics::ComputeConstraint(SimBufferPtr simBuffer){

    hipError_t hipError_t;

    glm::vec3 maxPosition = max_element_xyz(&simBuffer->m_positions) + glm::vec3(simBuffer->m_commonParam.radius);
    glm::vec3 minPosition = min_element_xyz(&simBuffer->m_positions) - glm::vec3(simBuffer->m_commonParam.radius);

    // Compute Constraints
    keComputeConstraint<<< 1 +  m_numParticles/256, 256>>>(dm_Data, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputeConstraint  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // Correct Positions
    keComputePositionCorrection<<< 1 +  m_numParticles/256, 256>>>(dm_Data, minPosition, maxPosition, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keComputePositionCorrection  %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

    // Update Corrected Positions
    keUpdateCorretedPosition<<< 1 +  m_numParticles/256, 256>>>(dm_Data, m_numParticles);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::ComputeConstraint-keUpdatePosition :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();

	hipMemcpy(simBuffer->m_positions.data(),      dm_Data.correctedPos,      m_numParticles*sizeof(glm::vec3),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
  	{
        printf("Error at HiPhysics::ComputeConstraint-memcpyDelPos :%s\n",hipGetErrorString(hipError_t));
        exit(1);
        return false;
  	}

    return true;
}

bool HiPhysics::UpdateVelPos(SimBufferPtr simBuffer){

    hipError_t hipError_t; // TODO : 맴버변수화 

    keUpdateVelPos<<< 1 +  m_numParticles/256, 256>>>(dm_Data, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysics::UpdateVelPos :%s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}

bool HiPhysics::GetRenderingVariable(SimBufferPtr simBuffer){

    hipError_t hipError_t;

    keGetRenderValues<<< 1 +  m_numParticles/256, 256>>>(dm_Data, m_numParticles);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        printf("Error at HiPhysicsPBD::keGetRenderValues %s\n",hipGetErrorString(hipError_t));
        exit(1);
    }
    hipDeviceSynchronize();
    return true;
}