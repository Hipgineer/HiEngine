#include "hip/hip_runtime.h"
#include "hiphysicsPBD.h"

#define PI  3.1415926535897932f
#define iPI 0.3183098861837906f
// __device__ void WendlandKernel(glm::vec3	dr)
// {
	
// }

__device__ float Poly6Kernel(float	H, float	R)
{
	float iH = 1.0f/H;
	//    res = 315    /(    64    *  PI *    H^9  ) * pow((H*H - R*R),3);
	float res = 315.0f * 0.015625f * iPI * pow(iH,9) * pow((H*H - R*R),3);
	if (R >= H) res = 0.0f;
	return res;
}

__device__ glm::vec3 SpikyGradKernel(float H, glm::vec3 dR)
{
	float iH = 1.0f/H;
	float R = length(dR);
	float iR = 1.0f/R;
	
	//    res = 45    /(   PI *    H^6  ) * pow((H - |dR|),2) dR / |dR|;
	glm::vec3 res = - 45.0f * iPI * powf(iH, 6) * powf((H - R), 2) * iR * dR;
	if (R >= H) res = glm::vec3(0.0f);
	if (R < 0.0001f) res = glm::vec3(0.0f);

	return res;
}

__global__ void keGetRenderValues(DeviceDataFluid dDataFluid,
								int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// dDataFluid.colorValues[idx] = length(dDataFluid.velocities[idx]);
		// dDataFluid.colorValues[idx] = static_cast<float>(dDataFluid.gridIndices[idx]);
		// dDataFluid.colorValues[idx] = dDataFluid.constraints[idx];
		dDataFluid.colorValues[idx] = dDataFluid.lambdas[idx];
		// dDataFluid.colorValues[idx] = length(dDataFluid.DeviceDataFluid[idx]);
	}
}

__global__ void keComputeGridID(DeviceDataFluid dDataFluid,
								glm::vec3 	v3MinPosition, 
								glm::vec3 	v3MaxPosition,
								int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{

		//TODO: compute outside of kernel function ;;;
		// float H = dDataFluid.commonParam->radius * 1.2f * 2.0f * 2.0f;
		float H = dDataFluid.commonParam->H;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dDataFluid.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dDataFluid.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dDataFluid.commonParam->radius) - v3MinPosition.z)/H)+1;

		int32_t tmpInt = static_cast<int32_t>((dDataFluid.correctedPos[idx].x - v3MinPosition.x -(dDataFluid.commonParam->radius))/H)
					+ (ix)*static_cast<int32_t>((dDataFluid.correctedPos[idx].z - v3MinPosition.z -(dDataFluid.commonParam->radius))/H)
					+ (ix)*(iz)*static_cast<int32_t>((dDataFluid.correctedPos[idx].y - v3MinPosition.y -(dDataFluid.commonParam->radius))/H);
		dDataFluid.gridIndices[idx] = tmpInt;
	}
}


__global__ void keCountParticlesInGrids(DeviceDataFluid dDataFluid,
								glm::vec3 	v3MinPosition, 
								glm::vec3 	v3MaxPosition,
								int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{

		//TODO: compute outside of kernel function ;;;
		float H = dDataFluid.commonParam->radius * 1.2f * 2.0f * 2.0f;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dDataFluid.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dDataFluid.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dDataFluid.commonParam->radius) - v3MinPosition.z)/H)+1;

		int32_t tmpInt = static_cast<int32_t>((dDataFluid.correctedPos[idx].x - v3MinPosition.x -(dDataFluid.commonParam->radius))/H)
					+ (ix)*static_cast<int32_t>((dDataFluid.correctedPos[idx].z - v3MinPosition.z -(dDataFluid.commonParam->radius))/H)
					+ (ix)*(iz)*static_cast<int32_t>((dDataFluid.correctedPos[idx].y - v3MinPosition.y -(dDataFluid.commonParam->radius))/H);
		atomicAdd(&dDataFluid.numPartInGrids[tmpInt], 1);
	}
}

/*
keComputeConstraint
1. 로컬 변수 정의
2. 그리드 순회
	3. 그리드 내 입자 순회
		4. 예외처리
		5. 실제물리식 (1에서 정의된 변수도 사용함)
6. 글로벌 변수 업데이트
 
 // Lambda 함수 시급
kernelVariable kv;
for_NearParticles(DeviceDataFluid dDataFluid, FunctionPointer PhysicsComputation(KV));

*/
// struct kernelVariables;
// __device__ void for_NearParticles(DeviceDataFluid dDataFluid, FunctionPointer Computeconstraint(kernelVariables KV))
// {
// 	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
// 	int32_t IID = idx;
// 	int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dDataFluid.commonParam->radius) - v3MinPosition.x)/KV.H)+1;
// 	int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dDataFluid.commonParam->radius) - v3MinPosition.y)/KV.H)+1;
// 	int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dDataFluid.commonParam->radius) - v3MinPosition.z)/KV.H)+1;
// 	for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
// 		for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
// 			for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
// 			{
// 				int32_t nearGridID = dDataFluid.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
// 				if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
// 				int32_t staJID = nearGridID == 0 ? 0 : dDataFluid.numPartInGrids[nearGridID-1];
// 				int32_t endJID = dDataFluid.numPartInGrids[nearGridID];
// 				for (int32_t JID = staJID; JID < endJID; ++JID)
// 				{
					
// 				}
// 			}

// 	KV.gradConstraintSqrSum += dot(KV.gradConstraintI,KV.gradConstraintI);
// 	KV.constraintI = KV.densityI*KV.iDensityI0 - 1.0f;
// }
struct kernelVariables {
	int32_t IID;
	float densityI0;
	float iDensityI0;
	float H;
};

struct computeConstraintKernelVariables : kernelVariables {
	float densityI = 0.0f;
	float constraintI = 0.0f;
	glm::vec3 gradConstraintI = glm::vec3(0.0f);
	float gradConstraintSqrSum = 0.0f;
};

inline __device__ void ComputeConstraint(int32_t &JID, 
									DeviceDataFluid &dDataFluid,
									computeConstraintKernelVariables &KV)
{
	int32_t IID = KV.IID;
	glm::vec3 displaceVectorIJ = dDataFluid.correctedPos[IID] - dDataFluid.correctedPos[JID];
	float distanceIJ = sqrt(glm::dot(displaceVectorIJ, displaceVectorIJ));

	if (distanceIJ < (KV.H * 0.5f))
	{
		float particleVolume = pow(2.0f * dDataFluid.commonParam->radius, 3);
		float kernelWeight = Poly6Kernel(0.5f * KV.H, distanceIJ);
		float densityJ0 = dDataFluid.phaseParam[dDataFluid.phases[JID]].density;

		KV.densityI += densityJ0 * particleVolume * kernelWeight;

		if (IID == JID) return;
		if (distanceIJ < KV.H * 0.00001f) return;

		glm::vec3 gradKernelWeight = SpikyGradKernel(0.5f * KV.H, displaceVectorIJ);
		glm::vec3 gradConstraintIJ = KV.iDensityI0 * densityJ0 * particleVolume * gradKernelWeight;
		KV.gradConstraintI += gradConstraintIJ;
		KV.gradConstraintSqrSum += dot(-gradConstraintIJ, -gradConstraintIJ);
	}
}

inline __device__ void ComputeConstraintToGlobal(DeviceDataFluid &dDataFluid,
										computeConstraintKernelVariables &KV)
{
	KV.gradConstraintSqrSum += dot(KV.gradConstraintI,KV.gradConstraintI);
	KV.constraintI = KV.densityI*KV.iDensityI0 - 1.0f;
	dDataFluid.lambdas[KV.IID] = - KV.constraintI / (KV.gradConstraintSqrSum + dDataFluid.commonParam->relaxationParameter);
}

__global__ void keComputeConstraint(DeviceDataFluid dDataFluid,
									glm::vec3 	v3MinPosition, 
									glm::vec3 	v3MaxPosition,
									int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// KernelVariables KV; 
		// for_NearParticles(ComputeConstraint, dDataFluid, KV); ==> 안에서 IID는 고정, JID는 내부에서 계산
		// dDataFluid.update;

		computeConstraintKernelVariables KV; 
		KV.IID					= idx;
		KV.densityI0			= dDataFluid.phaseParam[dDataFluid.phases[KV.IID]].density;
		KV.iDensityI0			= 1.0f/dDataFluid.phaseParam[dDataFluid.phases[KV.IID]].density;
		KV.H					= dDataFluid.commonParam->radius * 1.2f * 2.0f * 2.0f;

		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dDataFluid.commonParam->radius) - v3MinPosition.x)/KV.H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dDataFluid.commonParam->radius) - v3MinPosition.y)/KV.H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dDataFluid.commonParam->radius) - v3MinPosition.z)/KV.H)+1;
		for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
			for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
				for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
				{
					int32_t nearGridID = dDataFluid.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
					if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
					int32_t staJID = nearGridID == 0 ? 0 : dDataFluid.numPartInGrids[nearGridID-1];
					int32_t endJID = dDataFluid.numPartInGrids[nearGridID];
					for (int32_t JID = staJID; JID < endJID; ++JID)
					{
						// 모두 이런 형식일 것이므로!
						ComputeConstraint(JID, dDataFluid, KV);
					}
				}
		ComputeConstraintToGlobal(dDataFluid, KV);
	}
}

__global__ void keComputePositionCorrection(DeviceDataFluid dDataFluid,
											glm::vec3 	v3MinPosition, 
											glm::vec3 	v3MaxPosition,
											int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	int32_t IID = idx;

	if(idx < nParticles)
	{
		float density0 = dDataFluid.phaseParam[dDataFluid.phases[IID]].density;
		float iDensity0 = 1.0f/density0;
		dDataFluid.deltaPos[IID] = glm::vec3(0.0f);
		float H = dDataFluid.commonParam->radius * 1.2f * 2.0f * 2.0f;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dDataFluid.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dDataFluid.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dDataFluid.commonParam->radius) - v3MinPosition.z)/H)+1;
		
		for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
			for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
				for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
				{
					int32_t nearGridID = dDataFluid.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
					if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
					int32_t staJID = nearGridID == 0 ? 0 : dDataFluid.numPartInGrids[nearGridID-1];
					int32_t endJID = dDataFluid.numPartInGrids[nearGridID];
					for (int32_t JID = staJID; JID < endJID; ++JID)
					{
						glm::vec3 dr = dDataFluid.correctedPos[IID] - dDataFluid.correctedPos[JID];
						float dr2  = glm::dot(dr,dr);
						if ( dr2 < (H*H*0.25f) )
						{
							if (IID == JID) continue;
							if ( dr2 < (H*H*0.0000001f) ) continue;

							float volume = pow(2.0f*dDataFluid.commonParam->radius,3);
							float dlen   = sqrt(dr2);
							glm::vec3 gradKernel = SpikyGradKernel(0.5f*H, dr);
							
							float scorr = - dDataFluid.commonParam->scorrK * powf(Poly6Kernel(0.5f*H, dlen) / Poly6Kernel(0.5f*H, 0.5f*H*dDataFluid.commonParam->scorrDq),4.0f);

							dDataFluid.deltaPos[IID] += iDensity0 * ((dDataFluid.lambdas[IID] + dDataFluid.lambdas[JID])*0.5f + scorr) * dDataFluid.phaseParam[dDataFluid.phases[JID]].density * volume * gradKernel;							
						}
					}
				}
		// bool check = false;
		// if (dDataFluid.correctedPos[IID].x < dDataFluid.commonParam->AnalysisBox.minPoint.x)  check = true;
		// if (dDataFluid.correctedPos[IID].x > dDataFluid.commonParam->AnalysisBox.maxPoint.x)  check = true;
		// if (dDataFluid.correctedPos[IID].y < dDataFluid.commonParam->AnalysisBox.minPoint.y)  check = true;
		// if (dDataFluid.correctedPos[IID].y > dDataFluid.commonParam->AnalysisBox.maxPoint.y)  check = true;
		// if (dDataFluid.correctedPos[IID].z < dDataFluid.commonParam->AnalysisBox.minPoint.z)  check = true;
		// if (dDataFluid.correctedPos[IID].z > dDataFluid.commonParam->AnalysisBox.maxPoint.z)  check = true;
		// dDataFluid.deltaPos[IID] = -0.01f*dDataFluid.deltaPos[IID];
	}
}

__global__ void kePredictPosition(DeviceDataFluid dDataFluid, 
						 		int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dDataFluid.velocities[idx] += dDataFluid.commonParam->dt * dDataFluid.commonParam->gravity;
		dDataFluid.correctedPos[idx] = dDataFluid.positions[idx] + dDataFluid.commonParam->dt*dDataFluid.velocities[idx];
	}
}

__global__ void keUpdateCorretedPosition(DeviceDataFluid dDataFluid, 
						 				int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dDataFluid.correctedPos[idx] = dDataFluid.correctedPos[idx] + dDataFluid.deltaPos[idx];
		
		if (dDataFluid.correctedPos[idx].x < dDataFluid.commonParam->AnalysisBox.minPoint.x + dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].x = dDataFluid.commonParam->AnalysisBox.minPoint.x + dDataFluid.commonParam->radius;
		if (dDataFluid.correctedPos[idx].x > dDataFluid.commonParam->AnalysisBox.maxPoint.x - dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].x = dDataFluid.commonParam->AnalysisBox.maxPoint.x - dDataFluid.commonParam->radius;
		if (dDataFluid.correctedPos[idx].y < dDataFluid.commonParam->AnalysisBox.minPoint.y + dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].y = dDataFluid.commonParam->AnalysisBox.minPoint.y + dDataFluid.commonParam->radius;
		if (dDataFluid.correctedPos[idx].y > dDataFluid.commonParam->AnalysisBox.maxPoint.y - dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].y = dDataFluid.commonParam->AnalysisBox.maxPoint.y - dDataFluid.commonParam->radius;
		if (dDataFluid.correctedPos[idx].z < dDataFluid.commonParam->AnalysisBox.minPoint.z + dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].z = dDataFluid.commonParam->AnalysisBox.minPoint.z + dDataFluid.commonParam->radius;
		if (dDataFluid.correctedPos[idx].z > dDataFluid.commonParam->AnalysisBox.maxPoint.z - dDataFluid.commonParam->radius) dDataFluid.correctedPos[idx].z = dDataFluid.commonParam->AnalysisBox.maxPoint.z - dDataFluid.commonParam->radius;
	}
}

__global__ void keUpdateVelPos(DeviceDataFluid dDataFluid, 
						 		int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dDataFluid.velocities[idx] = (dDataFluid.correctedPos[idx] - dDataFluid.positions[idx])/dDataFluid.commonParam->dt;
		dDataFluid.positions[idx]  =  dDataFluid.correctedPos[idx];
		
		if (dDataFluid.positions[idx].x < dDataFluid.commonParam->AnalysisBox.minPoint.x + dDataFluid.commonParam->radius) dDataFluid.positions[idx].x = dDataFluid.commonParam->AnalysisBox.minPoint.x + dDataFluid.commonParam->radius;
		if (dDataFluid.positions[idx].x > dDataFluid.commonParam->AnalysisBox.maxPoint.x - dDataFluid.commonParam->radius) dDataFluid.positions[idx].x = dDataFluid.commonParam->AnalysisBox.maxPoint.x - dDataFluid.commonParam->radius;
		if (dDataFluid.positions[idx].y < dDataFluid.commonParam->AnalysisBox.minPoint.y + dDataFluid.commonParam->radius) dDataFluid.positions[idx].y = dDataFluid.commonParam->AnalysisBox.minPoint.y + dDataFluid.commonParam->radius;
		if (dDataFluid.positions[idx].y > dDataFluid.commonParam->AnalysisBox.maxPoint.y - dDataFluid.commonParam->radius) dDataFluid.positions[idx].y = dDataFluid.commonParam->AnalysisBox.maxPoint.y - dDataFluid.commonParam->radius;
		if (dDataFluid.positions[idx].z < dDataFluid.commonParam->AnalysisBox.minPoint.z + dDataFluid.commonParam->radius) dDataFluid.positions[idx].z = dDataFluid.commonParam->AnalysisBox.minPoint.z + dDataFluid.commonParam->radius;
		if (dDataFluid.positions[idx].z > dDataFluid.commonParam->AnalysisBox.maxPoint.z - dDataFluid.commonParam->radius) dDataFluid.positions[idx].z = dDataFluid.commonParam->AnalysisBox.maxPoint.z - dDataFluid.commonParam->radius;
	}
}

__global__ void kePredictPositionCloth(DeviceDataCloth dDataCloth, 
								DeviceSimParams dSimParam,
						 		int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(idx < nParticles)
	{
		if ((idx == 0) || (idx == 13))
			dDataCloth.velocities[idx] += glm::vec3(0.0f);
		else
			dDataCloth.velocities[idx] += dSimParam.commonParam->dt * dSimParam.commonParam->gravity;

		dDataCloth.correctedPos[idx] = dDataCloth.positions[idx] + dSimParam.commonParam->dt*dDataCloth.velocities[idx];

		// if (idx == 0)
		// {
		// 	dDataCloth.velocities[idx]  = glm::vec3(0.0);
		// 	dDataCloth.correctedPos[idx]= dDataCloth.positions[idx];
		// }
	}
}

__global__ void keComputeStretchCloth(DeviceDataCloth dDataCloth,
    								DeviceSimParams dSimParam,
									int64_t 	nStretchLines)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;

	if(idx < nStretchLines)
	{
		int32_t id0 = dDataCloth.stretchID[2*idx];
		int32_t id1 = dDataCloth.stretchID[2*idx + 1];

        glm::vec3 p0 = dDataCloth.correctedPos[id0];
        glm::vec3 p1 = dDataCloth.correctedPos[id1];
        
        glm::vec3 d = p1 - p0;
		glm::vec3 norm = glm::normalize(d);
		float len = glm::length(d);
		float len0= 2.0f * dSimParam.commonParam->radius;

        glm::vec3 dP = norm * 0.2f * (len - len0);
		
		if ((id0 == 0) || (id0 == 13))
		{

		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id0].x, dP.x);
			atomicAdd(&dDataCloth.deltaPos[id0].y, dP.y);
			atomicAdd(&dDataCloth.deltaPos[id0].z, dP.z);	
		}

		if ((id1 == 0) || (id1 == 13))
		{
			
		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id1].y, -dP.y);
			atomicAdd(&dDataCloth.deltaPos[id1].x, -dP.x);
			atomicAdd(&dDataCloth.deltaPos[id1].z, -dP.z);
		}


	}
}


__global__ void keComputeBendCloth(DeviceDataCloth dDataCloth,
    								DeviceSimParams dSimParam,
									int64_t 	nBendLines)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nBendLines)
	{

		int32_t id0 = dDataCloth.bendID[2*idx];
		int32_t id1 = dDataCloth.bendID[2*idx + 1];

        glm::vec3 p0 = dDataCloth.correctedPos[id0];
        glm::vec3 p1 = dDataCloth.correctedPos[id1];
        
        glm::vec3 d = p1 - p0;
		glm::vec3 norm = glm::normalize(d);
		float len = glm::length(d);
		float len0= 4.0f * dSimParam.commonParam->radius;


        glm::vec3 dP = norm * 0.2f * (len - len0);
		
		if ((id0 == 0) || (id0 == 13))
		{

		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id0].x, dP.x);
			atomicAdd(&dDataCloth.deltaPos[id0].y, dP.y);
			atomicAdd(&dDataCloth.deltaPos[id0].z, dP.z);	
		}

		if ((id1 == 0) || (id1 == 13))
		{
			
		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id1].y, -dP.y);
			atomicAdd(&dDataCloth.deltaPos[id1].x, -dP.x);
			atomicAdd(&dDataCloth.deltaPos[id1].z, -dP.z);
		}
	}
}


__global__ void keComputeShearCloth(DeviceDataCloth dDataCloth,
    								DeviceSimParams dSimParam,
									int64_t 	nShearLines)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nShearLines)
	{

		int32_t id0 = dDataCloth.shearID[2*idx];
		int32_t id1 = dDataCloth.shearID[2*idx + 1];

        glm::vec3 p0 = dDataCloth.correctedPos[id0];
        glm::vec3 p1 = dDataCloth.correctedPos[id1];
        
        glm::vec3 d = p1 - p0;
		glm::vec3 norm = glm::normalize(d);
		float len = glm::length(d);
		float len0= sqrt(2.0f) * 2.0f * dSimParam.commonParam->radius;

        glm::vec3 dP = norm * 0.2f * (len - len0);
		
		if ((id0 == 0) || (id0 == 13))
		{

		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id0].x, dP.x);
			atomicAdd(&dDataCloth.deltaPos[id0].y, dP.y);
			atomicAdd(&dDataCloth.deltaPos[id0].z, dP.z);	
		}

		if ((id1 == 0) || (id1 == 13))
		{
			
		}
		else
		{
			atomicAdd(&dDataCloth.deltaPos[id1].y, -dP.y);
			atomicAdd(&dDataCloth.deltaPos[id1].x, -dP.x);
			atomicAdd(&dDataCloth.deltaPos[id1].z, -dP.z);
		}
	}
}

__global__ void keUpdateCorretedPositionCloth(DeviceDataCloth dDataCloth, DeviceSimParams dSimParam, int64_t nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dDataCloth.correctedPos[idx] = dDataCloth.correctedPos[idx] + dDataCloth.deltaPos[idx];
		dDataCloth.deltaPos[idx] = glm::vec3(0.0f);
	}
}

__global__ void keUpdateVelPosCloth(DeviceDataCloth dDataCloth, 
    								DeviceSimParams dSimParam,
						 			int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// if (idx == 0) return;
		dDataCloth.velocities[idx] = (dDataCloth.correctedPos[idx] - dDataCloth.positions[idx])/dSimParam.commonParam->dt;
		dDataCloth.positions[idx]  =  dDataCloth.correctedPos[idx];
	}
}


__global__ void keGetRenderValuesCloth(DeviceDataCloth dDataCloth,
								int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// dDataCloth.colorValues[idx] = length(dDataCloth.velocities[idx]);
		// dDataCloth.colorValues[idx] = static_cast<float>(dDataCloth.gridIndices[idx]);
		// dDataCloth.colorValues[idx] = dDataCloth.constraints[idx];
		dDataCloth.colorValues[idx] = dDataCloth.velocities[idx].x;
		// dDataCloth.colorValues[idx] = length(dDataCloth.DeviceDataFluid[idx]);
	}
}
