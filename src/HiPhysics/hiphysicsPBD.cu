#include "hip/hip_runtime.h"
#include "hiphysicsPBD.h"

#define PI  3.1415926535897932f
#define iPI 0.3183098861837906f
// __device__ void WendlandKernel(glm::vec3	dr)
// {
	
// }

__device__ float Poly6Kernel(float	H, float	R)
{
	float iH = 1.0f/H;
	//    res = 315    /(    64    *  PI *    H^9  ) * pow((H*H - R*R),3);
	float res = 315.0f * 0.015625f * iPI * pow(iH,9) * pow((H*H - R*R),3);
	if (R >= H) res = 0.0f;
	return res;
}

__device__ glm::vec3 SpikyGradKernel(float H, glm::vec3 dR)
{
	float iH = 1.0f/H;
	float R = length(dR);
	float iR = 1.0f/R;
	
	//    res = 45    /(   PI *    H^6  ) * pow((H - |dR|),2) dR / |dR|;
	glm::vec3 res = - 45.0f * iPI * powf(iH, 6) * powf((H - R), 2) * iR * dR;
	if (R >= H) res = glm::vec3(0.0f);
	if (R < 0.0001f) res = glm::vec3(0.0f);

	return res;
}

__global__ void keGetRenderValues(DeviceData dData,
								int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// dData.colorValues[idx] = length(dData.velocities[idx]);
		// dData.colorValues[idx] = static_cast<float>(dData.gridIndices[idx]);
		// dData.colorValues[idx] = dData.constraints[idx];
		dData.colorValues[idx] = dData.lambdas[idx];
		// dData.colorValues[idx] = length(dData.deltaPos[idx]);
	}
}

__global__ void keComputeGridID(DeviceData dData,
								glm::vec3 	v3MinPosition, 
								glm::vec3 	v3MaxPosition,
								int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{

		//TODO: compute outside of kernel function ;;;
		// float H = dData.commonParam->radius * 1.2f * 2.0f * 2.0f;
		float H = dData.commonParam->H;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dData.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dData.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dData.commonParam->radius) - v3MinPosition.z)/H)+1;

		int32_t tmpInt = static_cast<int32_t>((dData.correctedPos[idx].x - v3MinPosition.x -(dData.commonParam->radius))/H)
					+ (ix)*static_cast<int32_t>((dData.correctedPos[idx].z - v3MinPosition.z -(dData.commonParam->radius))/H)
					+ (ix)*(iz)*static_cast<int32_t>((dData.correctedPos[idx].y - v3MinPosition.y -(dData.commonParam->radius))/H);
		dData.gridIndices[idx] = tmpInt;
	}
}


__global__ void keCountParticlesInGrids(DeviceData dData,
								glm::vec3 	v3MinPosition, 
								glm::vec3 	v3MaxPosition,
								int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{

		//TODO: compute outside of kernel function ;;;
		float H = dData.commonParam->radius * 1.2f * 2.0f * 2.0f;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dData.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dData.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dData.commonParam->radius) - v3MinPosition.z)/H)+1;

		int32_t tmpInt = static_cast<int32_t>((dData.correctedPos[idx].x - v3MinPosition.x -(dData.commonParam->radius))/H)
					+ (ix)*static_cast<int32_t>((dData.correctedPos[idx].z - v3MinPosition.z -(dData.commonParam->radius))/H)
					+ (ix)*(iz)*static_cast<int32_t>((dData.correctedPos[idx].y - v3MinPosition.y -(dData.commonParam->radius))/H);
		atomicAdd(&dData.numPartInGrids[tmpInt], 1);
	}
}

/*
keComputeConstraint
1. 로컬 변수 정의
2. 그리드 순회
	3. 그리드 내 입자 순회
		4. 예외처리
		5. 실제물리식 (1에서 정의된 변수도 사용함)
6. 글로벌 변수 업데이트
 
 // Lambda 함수 시급
kernelVariable kv;
for_NearParticles(DeviceData dData, FunctionPointer PhysicsComputation(KV));

*/
// struct kernelVariables;
// __device__ void for_NearParticles(DeviceData dData, FunctionPointer Computeconstraint(kernelVariables KV))
// {
// 	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
// 	int32_t IID = idx;
// 	int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dData.commonParam->radius) - v3MinPosition.x)/KV.H)+1;
// 	int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dData.commonParam->radius) - v3MinPosition.y)/KV.H)+1;
// 	int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dData.commonParam->radius) - v3MinPosition.z)/KV.H)+1;
// 	for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
// 		for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
// 			for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
// 			{
// 				int32_t nearGridID = dData.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
// 				if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
// 				int32_t staJID = nearGridID == 0 ? 0 : dData.numPartInGrids[nearGridID-1];
// 				int32_t endJID = dData.numPartInGrids[nearGridID];
// 				for (int32_t JID = staJID; JID < endJID; ++JID)
// 				{
					
// 				}
// 			}

// 	KV.gradConstraintSqrSum += dot(KV.gradConstraintI,KV.gradConstraintI);
// 	KV.constraintI = KV.densityI*KV.iDensityI0 - 1.0f;
// }
struct kernelVariables {
	int32_t IID;
	float densityI0;
	float iDensityI0;
	float H;
};

struct computeConstraintKernelVariables : kernelVariables {
	float densityI = 0.0f;
	float constraintI = 0.0f;
	glm::vec3 gradConstraintI = glm::vec3(0.0f);
	float gradConstraintSqrSum = 0.0f;
};

inline __device__ void ComputeConstraint(int32_t &JID, 
									DeviceData &dData,
									computeConstraintKernelVariables &KV)
{
	int32_t IID = KV.IID;
	glm::vec3 displaceVectorIJ = dData.correctedPos[IID] - dData.correctedPos[JID];
	float distanceIJ = sqrt(glm::dot(displaceVectorIJ, displaceVectorIJ));

	if (distanceIJ < (KV.H * 0.5f))
	{
		float particleVolume = pow(2.0f * dData.commonParam->radius, 3);
		float kernelWeight = Poly6Kernel(0.5f * KV.H, distanceIJ);
		float densityJ0 = dData.phaseParam[dData.phases[JID]].density;

		KV.densityI += densityJ0 * particleVolume * kernelWeight;

		if (IID == JID) return;
		if (distanceIJ < KV.H * 0.00001f) return;

		glm::vec3 gradKernelWeight = SpikyGradKernel(0.5f * KV.H, displaceVectorIJ);
		glm::vec3 gradConstraintIJ = KV.iDensityI0 * densityJ0 * particleVolume * gradKernelWeight;
		KV.gradConstraintI += gradConstraintIJ;
		KV.gradConstraintSqrSum += dot(-gradConstraintIJ, -gradConstraintIJ);
	}
}

inline __device__ void ComputeConstraintToGlobal(DeviceData &dData,
										computeConstraintKernelVariables &KV)
{
	KV.gradConstraintSqrSum += dot(KV.gradConstraintI,KV.gradConstraintI);
	KV.constraintI = KV.densityI*KV.iDensityI0 - 1.0f;
	dData.lambdas[KV.IID] = - KV.constraintI / (KV.gradConstraintSqrSum + dData.commonParam->relaxationParameter);
}

__global__ void keComputeConstraint(DeviceData dData,
									glm::vec3 	v3MinPosition, 
									glm::vec3 	v3MaxPosition,
									int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		// KernelVariables KV; 
		// for_NearParticles(ComputeConstraint, dData, KV); ==> 안에서 IID는 고정, JID는 내부에서 계산
		// dData.update;

		computeConstraintKernelVariables KV; 
		KV.IID					= idx;
		KV.densityI0			= dData.phaseParam[dData.phases[KV.IID]].density;
		KV.iDensityI0			= 1.0f/dData.phaseParam[dData.phases[KV.IID]].density;
		KV.H					= dData.commonParam->radius * 1.2f * 2.0f * 2.0f;

		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dData.commonParam->radius) - v3MinPosition.x)/KV.H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dData.commonParam->radius) - v3MinPosition.y)/KV.H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dData.commonParam->radius) - v3MinPosition.z)/KV.H)+1;
		for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
			for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
				for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
				{
					int32_t nearGridID = dData.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
					if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
					int32_t staJID = nearGridID == 0 ? 0 : dData.numPartInGrids[nearGridID-1];
					int32_t endJID = dData.numPartInGrids[nearGridID];
					for (int32_t JID = staJID; JID < endJID; ++JID)
					{
						// 모두 이런 형식일 것이므로!
						ComputeConstraint(JID, dData, KV);
					}
				}
		ComputeConstraintToGlobal(dData, KV);
	}
}

__global__ void keComputePositionCorrection(DeviceData dData,
											glm::vec3 	v3MinPosition, 
											glm::vec3 	v3MaxPosition,
											int64_t 	nParticles)
{
	int32_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	int32_t IID = idx;

	if(idx < nParticles)
	{
		float density0 = dData.phaseParam[dData.phases[IID]].density;
		float iDensity0 = 1.0f/density0;
		dData.deltaPos[IID] = glm::vec3(0.0f);
		float H = dData.commonParam->radius * 1.2f * 2.0f * 2.0f;
		int32_t ix = static_cast<int32_t>((v3MaxPosition.x - (dData.commonParam->radius) - v3MinPosition.x)/H)+1;
		int32_t iy = static_cast<int32_t>((v3MaxPosition.y - (dData.commonParam->radius) - v3MinPosition.y)/H)+1;
		int32_t iz = static_cast<int32_t>((v3MaxPosition.z - (dData.commonParam->radius) - v3MinPosition.z)/H)+1;
		
		for (int32_t yyy = -1 ; yyy < 2  ; ++yyy)
			for (int32_t zzz = -1 ; zzz < 2  ; ++zzz)
				for (int32_t xxx = -1 ; xxx < 2  ; ++xxx)
				{
					int32_t nearGridID = dData.gridIndices[idx] + xxx + ix*zzz + ix*iz*yyy;
					if ( (nearGridID < 0) || (nearGridID > ix*iy*iz-1) ) continue;
					int32_t staJID = nearGridID == 0 ? 0 : dData.numPartInGrids[nearGridID-1];
					int32_t endJID = dData.numPartInGrids[nearGridID];
					for (int32_t JID = staJID; JID < endJID; ++JID)
					{
						glm::vec3 dr = dData.correctedPos[IID] - dData.correctedPos[JID];
						float dr2  = glm::dot(dr,dr);
						if ( dr2 < (H*H*0.25f) )
						{
							if (IID == JID) continue;
							if ( dr2 < (H*H*0.0000001f) ) continue;

							float volume = pow(2.0f*dData.commonParam->radius,3);
							float dlen   = sqrt(dr2);
							glm::vec3 gradKernel = SpikyGradKernel(0.5f*H, dr);
							
							float scorr = - dData.commonParam->scorrK * powf(Poly6Kernel(0.5f*H, dlen) / Poly6Kernel(0.5f*H, 0.5f*H*dData.commonParam->scorrDq),4.0f);

							dData.deltaPos[IID] += iDensity0 * ((dData.lambdas[IID] + dData.lambdas[JID])*0.5f + scorr) * dData.phaseParam[dData.phases[JID]].density * volume * gradKernel;							
						}
					}
				}
		// bool check = false;
		// if (dData.correctedPos[IID].x < dData.commonParam->AnalysisBox.minPoint.x)  check = true;
		// if (dData.correctedPos[IID].x > dData.commonParam->AnalysisBox.maxPoint.x)  check = true;
		// if (dData.correctedPos[IID].y < dData.commonParam->AnalysisBox.minPoint.y)  check = true;
		// if (dData.correctedPos[IID].y > dData.commonParam->AnalysisBox.maxPoint.y)  check = true;
		// if (dData.correctedPos[IID].z < dData.commonParam->AnalysisBox.minPoint.z)  check = true;
		// if (dData.correctedPos[IID].z > dData.commonParam->AnalysisBox.maxPoint.z)  check = true;
		// dData.deltaPos[IID] = -0.01f*dData.deltaPos[IID];
	}
}

__global__ void kePredictPosition(DeviceData dData, 
						 		int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dData.velocities[idx] += dData.commonParam->dt * dData.commonParam->gravity;
		dData.correctedPos[idx] = dData.positions[idx] + dData.commonParam->dt*dData.velocities[idx];
	}
}


__global__ void keUpdateCorretedPosition(DeviceData dData, 
						 				int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dData.correctedPos[idx] = dData.correctedPos[idx] + dData.deltaPos[idx];
		
		if (dData.correctedPos[idx].x < dData.commonParam->AnalysisBox.minPoint.x) dData.correctedPos[idx].x = dData.commonParam->AnalysisBox.minPoint.x;
		if (dData.correctedPos[idx].x > dData.commonParam->AnalysisBox.maxPoint.x) dData.correctedPos[idx].x = dData.commonParam->AnalysisBox.maxPoint.x;
		if (dData.correctedPos[idx].y < dData.commonParam->AnalysisBox.minPoint.y) dData.correctedPos[idx].y = dData.commonParam->AnalysisBox.minPoint.y;
		if (dData.correctedPos[idx].y > dData.commonParam->AnalysisBox.maxPoint.y) dData.correctedPos[idx].y = dData.commonParam->AnalysisBox.maxPoint.y;
		if (dData.correctedPos[idx].z < dData.commonParam->AnalysisBox.minPoint.z) dData.correctedPos[idx].z = dData.commonParam->AnalysisBox.minPoint.z;
		if (dData.correctedPos[idx].z > dData.commonParam->AnalysisBox.maxPoint.z) dData.correctedPos[idx].z = dData.commonParam->AnalysisBox.maxPoint.z;
	}
}

__global__ void keUpdateVelPos(DeviceData dData, 
						 		int64_t 	nParticles)
{
	int64_t idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		dData.velocities[idx] = (dData.correctedPos[idx] - dData.positions[idx])/dData.commonParam->dt;
		dData.positions[idx]  =  dData.correctedPos[idx];
		
		if (dData.positions[idx].x < dData.commonParam->AnalysisBox.minPoint.x) dData.positions[idx].x = dData.commonParam->AnalysisBox.minPoint.x;
		if (dData.positions[idx].x > dData.commonParam->AnalysisBox.maxPoint.x) dData.positions[idx].x = dData.commonParam->AnalysisBox.maxPoint.x;
		if (dData.positions[idx].y < dData.commonParam->AnalysisBox.minPoint.y) dData.positions[idx].y = dData.commonParam->AnalysisBox.minPoint.y;
		if (dData.positions[idx].y > dData.commonParam->AnalysisBox.maxPoint.y) dData.positions[idx].y = dData.commonParam->AnalysisBox.maxPoint.y;
		if (dData.positions[idx].z < dData.commonParam->AnalysisBox.minPoint.z) dData.positions[idx].z = dData.commonParam->AnalysisBox.minPoint.z;
		if (dData.positions[idx].z > dData.commonParam->AnalysisBox.maxPoint.z) dData.positions[idx].z = dData.commonParam->AnalysisBox.maxPoint.z;
	}
}